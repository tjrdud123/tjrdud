
#include <hip/hip_runtime.h>
#include <cstdio>

// main program for the CPU: compiled by MS-VC++
int main(void) {
	// host-side data
	const int SIZE = 5;
	const int a[SIZE] = { 1, 2, 3, 4, 5 };
	int b[SIZE] = { 0, 0, 0, 0, 0 };
	// print source
	printf("a = {%d,%d,%d,%d,%d}\n", a[0], a[1], a[2], a[3], a[4]);
	// device-side data
	int* dev_a = 0;
	int* dev_b = 0;
	// allocate device memory
	hipMalloc((void**)&dev_a, SIZE * sizeof(int));
	hipMalloc((void**)&dev_b, SIZE * sizeof(int));
	// copy from host to device
	hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice); // dev_a = a;
	// copy from device to device
	hipMemcpy(dev_b, dev_a, SIZE * sizeof(int), hipMemcpyDeviceToDevice); // dev_b = dev_a;
	// copy from device to host
	hipMemcpy(b, dev_b, SIZE * sizeof(int), hipMemcpyDeviceToHost); // b = dev_b;
	// free device memory
	hipFree(dev_a);
	hipFree(dev_b);
	// print the result
	printf("b = {%d,%d,%d,%d,%d}\n", b[0], b[1], b[2], b[3], b[4]);
	// done
	return 0;
}

