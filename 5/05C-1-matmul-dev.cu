#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h> // for rand(), malloc(), free()
#include <windows.h> // for high-resolution performance counter

#if defined(NDEBUG)
#define CUDA_CHECK(x)	(x)
#else
#define CUDA_CHECK(x)	do {\
		(x); \
		hipError_t e = hipGetLastError(); \
		if (hipSuccess != e) { \
			printf("cuda failure \"%s\" at %s:%d\n", \
				hipGetErrorString(e), \
			     __FILE__, __LINE__); \
			exit(1); \
		} \
	} while (0)
#endif

#define WIDTH		(1 * 1024)	// total width is 1024*1024
#define	TILE_WIDTH	32		// block will be (TILE_WIDTH,TILE_WIDTH)
#define	GRID_WIDTH	(WIDTH / TILE_WIDTH)	// grid will be (GRID_WDITH,GRID_WDITH)


void genData(float* ptr, unsigned int size) {
	while (size--) {
		*ptr++ = (float)(rand() % 1000) / 1000.0F;
	}
}


__global__ void matmul(float* c, const float* a, const float* b, const int width) {
	// c[y][x] = sum_k a[y][k] * b[k][x]
	// c[y * WIDTH + x] = sum_k a[y*WIDTH + k] * b[k*WIDTH + x]
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0.0F;
	for (register int k = 0; k < width; ++k) {
		float lhs = a[y * width + k];
		float rhs = b[k * width + x];
		sum += lhs * rhs;
	}
	c[y * width + x] = sum;
}


int main(void) {
	float* pA = NULL;
	float* pB = NULL;
	float* pC = NULL;
	long long cntStart, cntEnd, freq;
	QueryPerformanceFrequency((LARGE_INTEGER*)(&freq));
	// malloc memories on the host-side
	pA = (float*)malloc(WIDTH * WIDTH * sizeof(float));
	pB = (float*)malloc(WIDTH * WIDTH * sizeof(float));
	pC = (float*)malloc(WIDTH * WIDTH * sizeof(float));
//	printf("pA, pB, pC = %#x %#x %#x\n", pA, pB, pC);
	// generate source data
	genData(pA, WIDTH * WIDTH);
	genData(pB, WIDTH * WIDTH);
	// CUDA: allocate device memory
	float* pAdev = NULL;
	float* pBdev = NULL;
	float* pCdev = NULL;
	CUDA_CHECK( hipMalloc((void**)&pAdev, WIDTH * WIDTH * sizeof(float)) );
	CUDA_CHECK( hipMalloc((void**)&pBdev, WIDTH * WIDTH * sizeof(float)) );
	CUDA_CHECK( hipMalloc((void**)&pCdev, WIDTH * WIDTH * sizeof(float)) );
//	printf("pAdev, pBdev, pCdev = %#x %#x %#x\n", pAdev, pBdev, pCdev);
	// CUDA: copy from host to device
	CUDA_CHECK( hipMemcpy(pAdev, pA, WIDTH * WIDTH * sizeof(float), hipMemcpyHostToDevice) );
	CUDA_CHECK( hipMemcpy(pBdev, pB, WIDTH * WIDTH * sizeof(float), hipMemcpyHostToDevice) );
	// start the timer
	QueryPerformanceCounter((LARGE_INTEGER*)(&cntStart)); // start the stop watch
	// CUDA: launch the kernel
	dim3 dimGrid(GRID_WIDTH, GRID_WIDTH, 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
	matmul <<< dimGrid, dimBlock>>>(pCdev, pAdev, pBdev, WIDTH);
	// end the timer
	QueryPerformanceCounter((LARGE_INTEGER*)(&cntEnd)); // end the stop watch
	CUDA_CHECK( hipPeekAtLastError() );
	printf("elapsed time = %f msec\n", (double)(cntEnd - cntStart) * 1000.0 / (double)(freq));
	// CUDA: copy from device to host
	CUDA_CHECK( hipMemcpy(pC, pCdev, WIDTH * WIDTH * sizeof(float), hipMemcpyDeviceToHost) );
	// print sample cases
	int i, j;
	i = 0; j = 0; printf("c[%4d][%4d] = %f\n", i, j, pC[i * WIDTH + j]);
	i = WIDTH / 2; j = WIDTH / 2; printf("c[%4d][%4d] = %f\n", i, j, pC[i * WIDTH + j]);
	i = WIDTH - 1; j = WIDTH - 1; printf("c[%4d][%4d] = %f\n", i, j, pC[i * WIDTH + j]);
	// CUDA: free the memory
	CUDA_CHECK( hipFree(pAdev) );
	CUDA_CHECK( hipFree(pBdev) );
	CUDA_CHECK( hipFree(pCdev) );
	// free the memory
	free(pA);
	free(pB);
	free(pC);
}

